#include "hip/hip_runtime.h"
#include "caesar_cipher.h"

#define MAX_MESSAGE_LENGTH 499
/**
 * Constant holder for message from text file
 */
__constant__ char const_message[MAX_MESSAGE_LENGTH];
/**
 * Setup the cuda timers
 *
 * start - the start marker
 * stop - the stop marker
 */
void setupTimer(hipEvent_t* start, hipEvent_t* stop){
    hipEventCreate(start);
    hipEventCreate(stop);
}
/**
 * Setup the cuda timers
 *
 * start - the start marker
 * stop - the stop marker
 * message - log message
 */
void logTime(hipEvent_t start, hipEvent_t stop, const char* message){
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("  %8s: %f\n", message, elapsed);
}
/**
 * Setup the cuda timers
 *
 * start - the start marker
 * stop - the stop marker
 */
void cleanTimer(hipEvent_t start, hipEvent_t stop){
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
/**
 * Kernel to modulus two arrays and put the remainder in a third array
 */
__global__
void caesarCipher(char * result, int offset, const unsigned int length) 
{
    extern __shared__ char shared[];
    if (threadIdx.x < length) {
        shared[threadIdx.x] = (const_message[threadIdx.x] + offset) % ASCII_MAX;
        __syncthreads();
        result[threadIdx.x] = shared[threadIdx.x];
    }
}
/**
 * Parse CLI arguments and store in an Options Structure
 *
 * argc - argument count
 * argv - argument variables
 * options - options structure
 */
void parseArgs(unsigned int argc, char** argv, Options* options){
    // set defaults
    options->filename = "cipher.txt";
    options->offset = 3;
    // first arg is file name
    if (argc >= 2) {
        options->filename = argv[1];
    }
    // second arg cipher offset
    if (argc >= 3) {
        options->offset = atoi(argv[2]);
    }
}
/**
 * Allocate the pinnable memory for fast access
 *
 * results - the pinned results
 * device - the device memory structure
 * size - the array size
 */
void allocate(Results* cipher, Results* decrypt, Device* device, unsigned int size) {
    hipMalloc((void **)&device->block, size);
    hipMalloc((void **)&device->encrypted, size);
    hipMalloc((void **)&device->decrypted, size);

    hipHostMalloc((void**)&cipher->output, size);
    hipHostMalloc((void**)&cipher->input, size);
    hipHostMalloc((void**)&decrypt->output, size);
    hipHostMalloc((void**)&decrypt->input, size);
    
}
/**
 * Allocate the pageable memory for fast access
 *
 * results - the pinned results
 * device - the device memory structure
 * size - the array size
 */
void allocatePageable(Results* cipher, Results* decrypt, Device* device, unsigned int size) {
    hipMalloc((void **)&device->block, size);
    hipMalloc((void **)&device->encrypted, size);
    hipMalloc((void **)&device->decrypted, size);

    cipher->output = (char*)malloc(size);
    cipher->input = (char*)malloc(size);
    decrypt->output = (char*)malloc(size);
    decrypt->input = (char*)malloc(size);
    
}
/**
 * Perform the encryption
 *
 * cipher - the string to encrypt structure
 * device - the device memory structure
 * size - the array size
 * offset - the cipher offset
 */
void encrypt(Results * cipher, Device * device, int size, int offset) {
    hipEvent_t start, stop;
    // setup timer
    setupTimer(&start, &stop);
    // transfer in
    hipEventRecord(start, 0);
    hipMemcpyToSymbol(HIP_SYMBOL(const_message), cipher->input, size);
    logTime(start, stop, CONST_TRANSFER_STRING);
    // execute kernel
    hipEventRecord(start, 0);
    caesarCipher<<<1,size, size>>>(device->block, offset, cipher->length);
    logTime(start, stop, KERNEL_STRING);
    // transfer out
    hipEventRecord(start, 0);
    hipMemcpy(cipher->output,
        device->block,
        size,
        hipMemcpyDeviceToHost);
    logTime(start, stop, TRANSFER_STRING);
    // clean up the timer
    cleanTimer(start, stop);
}
/**
 * Get the message from a file
 *
 * Options - the CLI options
 */
const char * getMessage(Options options) {
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    // open the file
    fp = fopen(options.filename, "r");  
    if (fp == NULL){
        printf("Couldn't find file %s\n", options.filename);
        exit(EXIT_FAILURE);
    }
    // read the first line
    getline(&line, &len, fp);
    // close the file
    fclose(fp);
    return line;
}
/**
 * Clean up the pinned memory allocations
 *
 * encrypted - results structure for encryption
 * decrypted - results structure for decryption
 * device - device structure
 */
void freePinned(Results encrypted, Results decrypted, Device device) {
    // free host side
    hipHostFree(encrypted.input);
    hipHostFree(encrypted.output);
    hipHostFree(decrypted.input);
    hipHostFree(decrypted.output);
    // free device side
    hipFree(device.encrypted);
    hipFree(device.decrypted);
    hipFree(device.block);
}
/**
 * Execute pinned strategy
 *
 * message - the message to encrypt
 * options - the cli options
 */
void execute_pinned(const char * message, Options options) {
    Results encrypted;
    Results decrypted;
    Device device;
    // store the string length
    encrypted.length = strlen(message);
    decrypted.length = encrypted.length;
    // calculate array size
    unsigned int size = sizeof(char) * encrypted.length;
    printf("  Pinned:\n");
    // allocate
    allocate(&encrypted, &decrypted, &device, size);
    memcpy(encrypted.input, message, sizeof(char) * encrypted.length);
    // perform encryption
    encrypt(&encrypted, &device, size, options.offset);
    printf("   Encrypted: %s\n", encrypted.output);
    // swap
    decrypted.input = encrypted.output;
    // perform decryption
    encrypt(&decrypted, &device, size, -options.offset);
    printf("   Decrypted: %s\n", decrypted.output);
    // free memory
    freePinned(encrypted, decrypted, device);
}
/**
 * Clean up the pageable memory allocations
 *
 * encrypted - results structure for encryption
 * decrypted - results structure for decryption
 * device - device structure
 */
void freePageable(Results encrypted, Results decrypted, Device device) {
    free(encrypted.input);
    free(encrypted.output);
    hipFree(device.encrypted);
    hipFree(device.decrypted);
    hipFree(device.block);
}
/**
 * Execute pageable strategy
 *
 * message - the message to encrypt
 * options - the cli options
 */
void execute_pageable(const char * message, Options options) {
    Results encrypted;
    Results decrypted;
    Device device;
    // store length
    encrypted.length = strlen(message);
    decrypted.length = encrypted.length;
    unsigned int size = sizeof(char) * encrypted.length;
    printf("  Pageable:\n");
    // allocate
    allocatePageable(&encrypted, &decrypted, &device, size);
    memcpy(encrypted.input, message, sizeof(char) * encrypted.length);
    // perform encryption
    encrypt(&encrypted, &device, size, options.offset);
    printf("   Encrypted: %s\n", encrypted.output);
    // swap
    decrypted.input = encrypted.output;
    // perform decryption
    encrypt(&decrypted, &device, size, -options.offset);
    printf("   Decrypted: %s\n", decrypted.output);
    // free memory
    freePageable(encrypted, decrypted, device);
}
/**
 * Main entrypoint
 */
int main(int argc, char** argv) {
    Options options;
    // parse cli args
    parseArgs(argc, argv, &options);
    // get the message from file
    const char * message = getMessage(options);
    printf("Message: %s\n", message);
    // execute pageable
    execute_pageable(message, options);
    // execute pinned
    execute_pinned(message, options);
    exit(EXIT_SUCCESS);
}