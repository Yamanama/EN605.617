#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// RANDOM_MAX ensures 0 <= result < 4 to meet the requirement
#define RANDOM_MAX 4

#define KERNEL_STRING "    Kernel"
#define TRANSFER_STRING "    Host Transfer"
#define CONST_TRANSFER_STRING "    Constant Transfer"

__constant__ unsigned int const_sequence[CONSTANT_MAX];
__constant__ unsigned int const_random[CONSTANT_MAX];

/**
 * Kernel to initial hiprand states
 * Derived from https://docs.nvidia.com/cuda/hiprand
 */
__global__
void initRandoms(unsigned int seed, hiprandState_t* states)
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    hiprand_init(seed, thread_idx, 0, &states[thread_idx]);
}
/**
 * Kernel to initialize randoms using hiprand states 
 * Derived from https://docs.nvidia.com/cuda/hiprand
 */
__global__
void randoms(hiprandState_t* states, unsigned int* result)
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    result[thread_idx] = hiprand(&states[thread_idx]) % RANDOM_MAX;
}
/**
 * Kernel to generate a sequence of numbers
 */
__global__
void sequence(unsigned int* result)
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    result[thread_idx] = thread_idx;
}
/**
 * Kernel to add two arrays and put the sums in a third array
 */
__global__
void add(unsigned int* result)
{
    extern __shared__ unsigned int shared[];
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    shared[thread_idx] = const_sequence[thread_idx] + const_random[thread_idx];
    __syncthreads();
    result[thread_idx] = shared[thread_idx];
}
/**
 * Kernel to subtract two arrays and put the differences in a third array
 */
__global__
void subtract(unsigned int * result)
{
    extern __shared__ unsigned int shared[];
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    shared[thread_idx] = const_sequence[thread_idx] - const_random[thread_idx];
    __syncthreads();
    result[thread_idx] = shared[thread_idx];
}
/**
 * Kernel to multiply two arrays and put the product in a third array
 */
__global__
void mult(unsigned int * result)
{
    extern __shared__ unsigned int shared[];
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    shared[thread_idx] = const_sequence[thread_idx] * const_random[thread_idx];
    __syncthreads();
    result[thread_idx] = shared[thread_idx];
}
/**
 * Kernel to modulus two arrays and put the remainder in a third array
 */
__global__
void mod(unsigned int * result)
{
    extern __shared__ unsigned int shared[];
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    shared[thread_idx] = const_sequence[thread_idx] % const_random[thread_idx];
    __syncthreads();
    result[thread_idx] = shared[thread_idx];
}
/**
 * Setup Timer
 *
 * start - start marker
 * stop - stop marker
 */
void setupTimer(hipEvent_t* start, hipEvent_t* stop){
    hipEventCreate(start);
    hipEventCreate(stop);
}
/**
 * Log the computed time
 * 
 * start - start marker
 * stop - stop marker
 * message - log message
 */
void logTime(hipEvent_t start, hipEvent_t stop, const char* message){
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("  %8s: %f\n", message, elapsed);
}
/**
 *  Clean up memory for timers
 *
 * start - start marker
 * stop - stop marker
 */
void cleanTimer(hipEvent_t start, hipEvent_t stop){
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
/**
 * Generate Sequence Array
 *
 * results - results structure
 * device - device structure
 * options - cli structure
 * message - log message
 */
void generateSequence(Results* results, 
                      Device device,
                      Options options, 
                      const char* message) 
{
    // markers
    hipEvent_t start, stop;
    printf("%sSequence:\n", message);
    // setup timer
    setupTimer(&start, &stop);
    // mart start
    hipEventRecord(start, 0);
    // execute the sequence kernel
    sequence<<<options.totalBlocks, options.blockSize>>>(device.block);
    // log time
    logTime(start, stop, KERNEL_STRING);
    hipEventRecord(start, 0);
    // store generated sequences
    hipMemcpy(results->sequence,
               device.block,
               options.arraySize,
               hipMemcpyDeviceToHost);
    logTime(start, stop, TRANSFER_STRING);
    // store in constant
    hipEventRecord(start, 0);
    hipMemcpyToSymbol(HIP_SYMBOL(const_sequence), device.block, options.arraySize);
    //log time
    logTime(start, stop, CONST_TRANSFER_STRING);
    //cleanup timer
    cleanTimer(start, stop);
}
/**
 * Generate Random Array
 *
 * results - results structure
 * device - device structure
 * options - cli structure
 * message - log message
 */
void generateRandom(Results* results, 
                    Device device, 
                    Options options, 
                    const char* message) 
{
    // markers
    hipEvent_t start, stop;
    printf("%sRandom:\n", message);
    // setup timer
    setupTimer(&start, &stop);
    hipMalloc((void**)&device.states,
                options.totalThreads * sizeof(hiprandState_t));
    // mark time
    hipEventRecord(start, 0);
    // execute the sequence kernel
    initRandoms<<<options.totalBlocks, options.blockSize>>>(time(0), 
                                                            device.states);
    randoms<<<options.totalBlocks, options.blockSize>>>(device.states, 
                                                        device.block);
    // log time
    logTime(start, stop, KERNEL_STRING);
    hipEventRecord(start, 0);
    // store generated sequences
    hipMemcpy(results->random,
               device.block,
               options.arraySize,
               hipMemcpyDeviceToHost);
    // log time
    logTime(start, stop, TRANSFER_STRING);
    hipEventRecord(start, 0);
    // store in constant
    hipMemcpyToSymbol(HIP_SYMBOL(const_random), device.block, options.arraySize);
    // log time
    logTime(start, stop, CONST_TRANSFER_STRING);
    // clean timer
    cleanTimer(start, stop);
}
/**
 * Perform addition
 *
 * results - results structure
 * device - device structure
 * options - cli structure
 */
void performAdd(Results* results, Device device, Options options)
{
    // markers
    hipEvent_t start, stop;
    printf("  Adding\n");
    // set up timer
    setupTimer(&start, &stop);
    // mark time
    hipEventRecord(start, 0);
    // calculate the necessary shared size
    const unsigned int shared_size = sizeof(unsigned int) * options.arraySize;
    // execute the kernel
    add<<<options.totalBlocks, options.blockSize, shared_size>>>(device.block);
    // log time
    logTime(start, stop, KERNEL_STRING);
    hipEventRecord(start, 0);
    // store generated sequences
    hipMemcpy(results->sum, 
               device.block, 
               options.arraySize,
               hipMemcpyDeviceToHost);
    // log time
    logTime(start, stop, TRANSFER_STRING);
    // clean up
    cleanTimer(start, stop);
}
/**
 * Perform subtraction
 *
 * results - results structure
 * device - device structure
 * options - cli structure
 */
void performSubtract(Results* results, Device device, Options options)
{
    // markers
    hipEvent_t start, stop;
    printf("  Subtracting\n");
    setupTimer(&start, &stop);
    // mark time
    hipEventRecord(start, 0);
    // calculate the necessary shared size
    const unsigned int shared_size = sizeof(unsigned int) * options.arraySize;
    // execute the kernel
    subtract<<<options.totalBlocks, options.blockSize, shared_size>>>(device.block);
    // log time
    logTime(start, stop, KERNEL_STRING);
    hipEventRecord(start, 0);
    // store generated sequences
    hipMemcpy(results->difference, 
               device.block, 
               options.arraySize,
               hipMemcpyDeviceToHost);
    // log time
    logTime(start, stop, TRANSFER_STRING);
    // clean up
    cleanTimer(start, stop);
}
/**
 * Perform modulus
 *
 * results - results structure
 * device - device structure
 * options - cli structure
 */
void performModulus(Results* results, Device device, Options options)
{
    // marker
    hipEvent_t start, stop;
    printf("  Modulus\n");
    setupTimer(&start, &stop);
    // mark time
    hipEventRecord(start, 0);
    // calculate the necessary shared size
    const unsigned int shared_size = sizeof(unsigned int) * options.arraySize;
    // execute the sequence kernel
    mod<<<options.totalBlocks, options.blockSize, shared_size>>>(device.block);
    // log time
    logTime(start, stop, KERNEL_STRING);
    hipEventRecord(start, 0);
    // store generated sequences
    hipMemcpy(results->modulus, 
               device.block, 
               options.arraySize,
               hipMemcpyDeviceToHost);
    // log time
    logTime(start, stop, TRANSFER_STRING);
    // clean up
    cleanTimer(start, stop);
}
/**
 * Perform multiple
 *
 * results - results structure
 * device - device structure
 * options - cli structure
 */
void performMult(Results* results, Device device, Options options)
{
    // array sizes
    hipEvent_t start, stop;
    printf("  Multiplying\n");
    setupTimer(&start, &stop);
    // mark time
    hipEventRecord(start, 0);
    // calculate the necessary shared size
    const unsigned int shared_size = sizeof(unsigned int) * options.arraySize;
    // execute the kernel
    mult<<<options.totalBlocks, options.blockSize, shared_size>>>(device.block);
    // log time
    logTime(start, stop, KERNEL_STRING);
    hipEventRecord(start, 0);
    // store generated sequences
    hipMemcpy(results->product, 
               device.block, 
               options.arraySize,
               hipMemcpyDeviceToHost);
    // log time
    logTime(start, stop, TRANSFER_STRING);
    // clean up timers
    cleanTimer(start, stop);
}
/**
 * Perform addition
 *
 * results - results structure
 * device - device structure
 * options - cli structure
 * message - log message
 */
void perform(Results results, 
             Device device, 
             Options options, 
             const char * message)
{
    printf("%s\n  Generate:\n", message);
    // generate
    generateSequence(&results, device, options, "    ");
    generateRandom(&results, device, options, "    ");
    // perform
    performAdd(&results, device, options);
    performSubtract(&results, device, options);
    performMult(&results, device, options);
    performModulus(&results, device, options);
}
