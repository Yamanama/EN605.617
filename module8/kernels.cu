#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>


/**
 * Setup Timer
 *
 * start - start marker
 * stop - stop marker
 */
 void setupTimer(hipEvent_t* start, hipEvent_t* stop){
    hipEventCreate(start);
    hipEventCreate(stop);
}
/**
 * Log the computed time
 * 
 * start - start marker
 * stop - stop marker
 * message - log message
 */
void logTime(hipEvent_t start, hipEvent_t stop, const char* message){
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("  %8s: %f\n", message, elapsed);
}
/**
 *  Clean up memory for timers
 *
 * start - start marker
 * stop - stop marker
 */
void cleanTimer(hipEvent_t start, hipEvent_t stop){
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

/**
 * Generate a random matrix on the device
 *
 * array - the array
 * rows - the rows
 * columns - the columns
 */
void generateRandomMatrix(Matrix array, int rows, int columns) {
    // generator
    hiprandGenerator_t numberGenerator;
    hiprandCreateGenerator(&numberGenerator, HIPRAND_RNG_PSEUDO_DEFAULT);
    // timer
    hipEvent_t start, stop;
    setupTimer(&start, &stop);
    // generate
    hipEventRecord(start, 0);
    hiprandSetPseudoRandomGeneratorSeed(numberGenerator, 
                                       (unsigned long long) clock());
    hiprandGenerateUniform(numberGenerator, array.device, rows * columns);
    logTime(start, stop, "  Random Array Generation");
    hipEventRecord(start, 0);
    // transfer
    hipMemcpy(array.host,
        array.device,
        rows * columns * sizeof(float),
        hipMemcpyDeviceToHost);
    logTime(start, stop, "  Transfer to host");
    // clean up timers
    cleanTimer(start, stop);
}

/**
 * Generate a random vector on the device
 *
 * array - the array
 * rows - the rows
 * columns - the columns
 */
 void generateRandomVectors(Dot dot, int size) {
    // generator
    hiprandGenerator_t numberGenerator;
    hiprandCreateGenerator(&numberGenerator, HIPRAND_RNG_PSEUDO_DEFAULT);
    // timer
    hipEvent_t start, stop;
    setupTimer(&start, &stop);
    // generate
    hipEventRecord(start, 0);
    hiprandSetPseudoRandomGeneratorSeed(numberGenerator, 
                                       (unsigned long long) clock());
    hiprandGenerateUniform(numberGenerator, (float*)dot.vectorOne, size);
    hiprandGenerateUniform(numberGenerator, (float*)dot.vectorTwo, size);
    logTime(start, stop, "  Random Vectors Generation");
    // clean up timers
    cleanTimer(start, stop);
}

/**
 * Compute a dot product of two vectors
 * 
 * dot - the dot structure
 */
void computeDotProduct(Dot dot) {
    // timer
    hipEvent_t start, stop;
    setupTimer(&start, &stop);
    // handler
    hipblasHandle_t handler;
    hipblasCreate(&handler);
    // calculate
    hipEventRecord(start, 0);
    hipblasDdot(handler, DOT_LENGTH, dot.vectorOne, 1, dot.vectorTwo, 1, dot.result);
    logTime(start, stop, "  Random Vectors Generation");
    double result;
    hipMemcpy(&result, dot.result, sizeof(double), hipMemcpyDeviceToHost);
    // clean up timers
    printf("  Dot Product Result: %f\n", result);
    cleanTimer(start, stop);
    hipblasDestroy(handler);
}

/**
 * Perform addition
 *
 * results - results structure
 * device - device structure
 * options - cli structure
 * message - log message
 */
 void perform(Matrix A, Matrix B, Matrix C, const char * message) {
    printf("%s\n  Computing Product:\n", message);
    // setup timer
    hipEvent_t start, stop;
    setupTimer(&start, &stop);
    // temps
    const float alpha = 1.0f;
    const float beta  = 0.0f;
    // handler
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // calculate
    hipEventRecord(start, 0);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                A_ROWS, B_COLUMNS, A_COLUMNS, &alpha, 
                A.device, A_ROWS, B.device, A_COLUMNS, &beta, 
                C.device, A_ROWS);
    logTime(start, stop, "  Matrices Product Calculation");
    hipEventRecord(start, 0);
    // transfer
    hipMemcpy(C.host,
               C.device,
               C_ROWS * C_COLUMNS * sizeof(float),
               hipMemcpyDeviceToHost);
    logTime(start, stop, "  Transfer to host");
    hipblasDestroy(handle);
 }