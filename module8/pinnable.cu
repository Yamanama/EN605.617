
#include "common.cu"
#include "kernels.cu"
/**
 * Allocate the pinnable memory for fast access
 *
 * A - The first matrix
 * B - The second matrix
 * C - The product matrix
 */
void allocate(Matrix* A, Matrix* B, Matrix* C, Dot* dot) {
    // compute sizes
    unsigned int aSize = A_ROWS * A_COLUMNS * sizeof(float);
    unsigned int bSize = B_ROWS * B_COLUMNS * sizeof(float);
    unsigned int cSize = C_ROWS * C_COLUMNS * sizeof(float);
    unsigned int dotSize = sizeof(double) * (size_t)DOT_LENGTH;
    // device allocate
    hipMalloc((void**)&A->device, aSize);
    hipMalloc((void**)&B->device, bSize);
    hipMalloc((void**)&C->device, cSize);
    hipMalloc((void**)&dot->vectorOne, dotSize);
    hipMalloc((void**)&dot->vectorTwo, dotSize);
    hipMalloc((void**)&dot->result, sizeof(double));
    // host allocate
    hipHostMalloc((void**)&A->host, aSize);
    hipHostMalloc((void**)&B->host, bSize);
    hipHostMalloc((void**)&C->host, cSize);
    
}

/**
 * Cleanup memory
 *
 * A - the first matrix
 * B - the second matrix
 * C - the product matrix
 * dot - the dot product structure 
 */
 void freeMemory(Matrix A, Matrix B, Matrix C, Dot dot) {
    // free host
    hipHostFree(A.host);
    hipHostFree(B.host);
    hipHostFree(C.host);
    // free device
    hipFree(A.device);
    hipFree(B.device);
    hipFree(C.device);
    hipFree(dot.vectorOne);
    hipFree(dot.vectorTwo);
    hipFree(dot.result);
}

/**
 * Main entrypoint
 */
int main(int argc, char** argv) {
    Matrix A, B, C;
    Dot dot;
    // print device info
    printDevice();
    // allocate memory
    allocate(&A, &B, &C, &dot);
    // generate
    printf("Pinnable\n");
    printf("  A %dx%d Array Generation:\n", A_ROWS, A_COLUMNS);
    generateRandomMatrix(A, A_ROWS, A_COLUMNS);
    printf("  B %dx%d Array Generation:\n", B_ROWS, B_COLUMNS);
    generateRandomMatrix(B, B_ROWS, B_COLUMNS);
    // calculate
    perform(A, B, C, "Executing");
    // print
    print_matrices(A, B, C);
    // calculate a dot product
    printf("  Dot Product:\n");
    generateRandomVectors(dot, DOT_LENGTH);
    computeDotProduct(dot);
    // free
    freeMemory(A, B, C, dot);
}




