#include "common.cu"
#include "kernels.cu"
/**
 * Allocate the pinnable memory for fast access
 *
 * results - the pinned results
 * device - the device memory structure
 * options - cli options 
 */
void allocate(Results* results, Device* device, Options options) {
    unsigned int size = (sizeof(unsigned int) * options.totalThreads);
    hipMalloc((void**)&device->states,
                options.totalThreads * sizeof(hiprandState_t));
    hipMalloc((void **)&device->block, size);
    hipMalloc((void **)&device->sequence, size);
    hipMalloc((void **)&device->random, size);

    hipHostMalloc((void**)&results->sequence, options.arraySize);
    hipHostMalloc((void**)&results->random, options.arraySize);
    hipHostMalloc((void**)&results->sum, options.arraySize);
    hipHostMalloc((void**)&results->product, options.arraySize);
    hipHostMalloc((void**)&results->difference, options.arraySize);
    hipHostMalloc((void**)&results->modulus, options.arraySize);
}

/**
 * Cleanup memory
 *
 * results - the results
 * device - the device memory structure 
 */
 void freeMemory(Results results, Device device) {
    // free host
    hipHostFree(results.sequence);
    hipHostFree(results.random);
    hipHostFree(results.sum);
    hipHostFree(results.difference);
    hipHostFree(results.product);
    hipHostFree(results.modulus);
    // free device
    hipFree(device.states);
    hipFree(device.block);
    hipFree(device.sequence);
    hipFree(device.random);
}

/**
 * Main entrypoint
 */
int main(int argc, char** argv) {
    Options options;
    Results results;
    Device device;
    // parse cli args
    parseArgs(argc, argv, &options);
    // print device info
    printDevice();
    // allocate memory
    allocate(&results, &device, options);
    perform(results, device, options, "Pinnable");
    // printResults(results, options);
    freeMemory(results, device);
}




