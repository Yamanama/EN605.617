#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define KERNEL_LOOP 1024
/**
 * Setup the cuda timers
 *
 * start - the start marker
 * stop - the stop marker
 */
 void setupTimer(hipEvent_t* start, hipEvent_t* stop){
        hipEventCreate(start);
        hipEventCreate(stop);
    }
    /**
     * Setup the cuda timers
     *
     * start - the start marker
     * stop - the stop marker
     * message - log message
     */
    void logTime(hipEvent_t start, hipEvent_t stop, const char* message){
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float elapsed;
        hipEventElapsedTime(&elapsed, start, stop);
        printf("  %8s: %f\n", message, elapsed);
    }
    /**
     * Setup the cuda timers
     *
     * start - the start marker
     * stop - the stop marker
     */
    void cleanTimer(hipEvent_t start, hipEvent_t stop){
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
__host__ void wait_exit(void)
{
        char ch;

        printf("\nPress any key to exit");
        ch = getchar();
}

__host__ void generate_rand_data(unsigned int * host_data_ptr)
{
        for(unsigned int i=0; i < KERNEL_LOOP; i++)
        {
                host_data_ptr[i] = (unsigned int) rand();
        }
}

__global__ void test_gpu_register(unsigned int * const data, const unsigned int num_elements)
{
        const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if(tid < num_elements)
        {
                unsigned int d_tmp = data[tid];
                d_tmp = d_tmp * 2;
                data[tid] = d_tmp;
        }
}

__host__ void gpu_kernel(void)
{
        const unsigned int num_elements = KERNEL_LOOP;
        const unsigned int num_threads = KERNEL_LOOP;
        const unsigned int num_blocks = num_elements/num_threads;
        const unsigned int num_bytes = num_elements * sizeof(unsigned int);

        unsigned int * data_gpu;

        unsigned int host_packed_array[num_elements];
        unsigned int host_packed_array_output[num_elements];
        hipEvent_t start, stop;
        // setup timer
        setupTimer(&start, &stop);
        hipMalloc(&data_gpu, num_bytes);

        generate_rand_data(host_packed_array);

        hipMemcpy(data_gpu, host_packed_array, num_bytes,hipMemcpyHostToDevice);
        hipEventRecord(start, 0);
        test_gpu_register <<<num_blocks, num_threads>>>(data_gpu, num_elements);
        logTime(start, stop, "Time: ");
        hipDeviceSynchronize();        // Wait for the GPU launched work to complete
        hipGetLastError();

        hipMemcpy(host_packed_array_output, data_gpu, num_bytes,hipMemcpyDeviceToHost);

        for (int i = 0; i < num_elements; i++){
                printf("Input value: %x, device output: %x\n",host_packed_array[i], host_packed_array_output[i]);
        }

        hipFree((void* ) data_gpu);
        hipDeviceReset();
        wait_exit();
}

void execute_host_functions()
{

}

void execute_gpu_functions()
{
	gpu_kernel();
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {
	execute_host_functions();
	execute_gpu_functions();

	return EXIT_SUCCESS;
}
