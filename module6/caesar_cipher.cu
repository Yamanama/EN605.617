#include "hip/hip_runtime.h"
#include "caesar_cipher.h"

/**
 * Setup the cuda timers
 *
 * start - the start marker
 * stop - the stop marker
 */
void setupTimer(hipEvent_t* start, hipEvent_t* stop){
    hipEventCreate(start);
    hipEventCreate(stop);
}
/**
 * Setup the cuda timers
 *
 * start - the start marker
 * stop - the stop marker
 * message - log message
 */
void logTime(hipEvent_t start, hipEvent_t stop, const char* message){
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("  %8s: %f\n", message, elapsed);
}
/**
 * Setup the cuda timers
 *
 * start - the start marker
 * stop - the stop marker
 */
void cleanTimer(hipEvent_t start, hipEvent_t stop){
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
/**
 * Kernel to modulus two arrays and put the remainder in a third array
 */
__global__
void caesarCipher(char * result, int offset) 
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    char d_tmp = (result[thread_idx] + offset) % ASCII_MAX;
    result[thread_idx] = d_tmp;
}
/**
 * Parse CLI arguments and store in an Options Structure
 *
 * argc - argument count
 * argv - argument variables
 * options - options structure
 */
void parseArgs(unsigned int argc, char** argv, Options* options){
    // set defaults
    options->filename = "cipher.txt";
    options->offset = 3;
    // first arg is file name
    if (argc >= 2) {
        options->filename = argv[1];
    }
    // second arg cipher offset
    if (argc >= 3) {
        options->offset = atoi(argv[2]);
    }
}
/**
 * Allocate the pinnable memory for fast access
 *
 * results - the pinned results
 * device - the device memory structure
 * size - the array size
 */
void allocate(Results* cipher, Results* decrypt, Device* device, unsigned int size) {
    hipMalloc((void **)&device->block, size);
    hipMalloc((void **)&device->encrypted, size);
    hipMalloc((void **)&device->decrypted, size);

    hipHostMalloc((void**)&cipher->output, size);
    hipHostMalloc((void**)&cipher->input, size);
    hipHostMalloc((void**)&decrypt->output, size);
    hipHostMalloc((void**)&decrypt->input, size);
    
}
/**
 * Allocate the pageable memory for fast access
 *
 * results - the pinned results
 * device - the device memory structure
 * size - the array size
 */
void allocatePageable(Results* cipher, Results* decrypt, Device* device, unsigned int size) {
    hipMalloc((void **)&device->block, size);
    hipMalloc((void **)&device->encrypted, size);
    hipMalloc((void **)&device->decrypted, size);

    cipher->output = (char*)malloc(size);
    cipher->input = (char*)malloc(size);
    decrypt->output = (char*)malloc(size);
    decrypt->input = (char*)malloc(size);
    
}
/**
 * Perform the encryption
 *
 * cipher - the string to translate structure
 * device - the device memory structure
 * size - the array size
 * offset - the cipher offset
 */
void translate(Results * cipher, Device * device, int size, int offset) {
    hipEvent_t start, stop;
    // setup timer
    setupTimer(&start, &stop);
    // transfer in
    hipEventRecord(start, 0);
    hipMemcpy(device->block, cipher->input, size, hipMemcpyHostToDevice);
    logTime(start, stop, TRANSFER_STRING);
    // execute kernel
    hipEventRecord(start, 0);
    caesarCipher<<<64,64>>>(device->block, offset);
    logTime(start, stop, KERNEL_STRING);
    // wait for the work to be done
    hipDeviceSynchronize();
    // check errors
    hipGetLastError();
    // transfer out
    hipEventRecord(start, 0);
    // copy it out
    hipMemcpy(cipher->output,
        device->block,
        size,
        hipMemcpyDeviceToHost);
    logTime(start, stop, TRANSFER_STRING);
    // clean up the timer
    cleanTimer(start, stop);
}
/**
 * Get the message from a file
 *
 * Options - the CLI options
 */
const char * getMessage(Options options) {
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    // open the file
    fp = fopen(options.filename, "r");  
    if (fp == NULL){
        printf("Couldn't find file %s\n", options.filename);
        exit(EXIT_FAILURE);
    }
    // read the first line
    getline(&line, &len, fp);
    // close the file
    fclose(fp);
    return line;
}
/**
 * Clean up the pinned memory allocations
 *
 * encrypted - results structure for encryption
 * decrypted - results structure for decryption
 * device - device structure
 */
void freePinned(Results encrypted, Results decrypted, Device device) {
    // free host side
    hipHostFree(encrypted.input);
    hipHostFree(encrypted.output);
    hipHostFree(decrypted.input);
    hipHostFree(decrypted.output);
    // free device side
    hipFree(device.encrypted);
    hipFree(device.decrypted);
    hipFree(device.block);
}
/**
 * Execute pinned strategy
 *
 * message - the message to translatee
 * options - the cli options
 */
void execute_pinned(const char * message, Options options) {
    Results encrypted;
    Results decrypted;
    Device device;
    // store the string length
    encrypted.length = strlen(message);
    // calculate array size
    unsigned int size = sizeof(char) * encrypted.length;
    printf("  Pinned:\n");
    // allocate
    allocate(&encrypted, &decrypted, &device, size);
    memcpy(encrypted.input, message, sizeof(char) * encrypted.length);
    // perform encryption
    translate(&encrypted, &device, size, options.offset);
    printf("   Encrypted: %s\n", encrypted.output);
    // swap
    decrypted.input = encrypted.output;
    // perform decryption
    translate(&decrypted, &device, size, -options.offset);
    printf("   Decrypted: %s\n", decrypted.output);
    // free memory
    freePinned(encrypted, decrypted, device);
}
/**
 * Clean up the pageable memory allocations
 *
 * encrypted - results structure for encryption
 * decrypted - results structure for decryption
 * device - device structure
 */
void freePageable(Results encrypted, Results decrypted, Device device) {
    free(encrypted.input);
    free(encrypted.output);
    hipFree(device.encrypted);
    hipFree(device.decrypted);
    hipFree(device.block);
}
/**
 * Execute pageable strategy
 *
 * message - the message to translatee
 * options - the cli options
 */
void execute_pageable(const char * message, Options options) {
    Results encrypted;
    Results decrypted;
    Device device;
    // store length
    encrypted.length = strlen(message);
    unsigned int size = sizeof(char) * encrypted.length;
    printf("  Pageable:\n");
    // allocate
    allocatePageable(&encrypted, &decrypted, &device, size);
    memcpy(encrypted.input, message, sizeof(char) * encrypted.length);
    // perform encryption
    translate(&encrypted, &device, size, options.offset);
    printf("   Encrypted: %s\n", encrypted.output);
    // swap
    decrypted.input = encrypted.output;
    // perform decryption
    translate(&decrypted, &device, size, -options.offset);
    printf("   Decrypted: %s\n", decrypted.output);
    // free memory
    freePageable(encrypted, decrypted, device);
}
/**
 * Main entrypoint
 */
int main(int argc, char** argv) {
    Options options;
    // parse cli args
    parseArgs(argc, argv, &options);
    // get the message from file
    const char * message = getMessage(options);
    printf("Message: %s\n", message);
    // execute pageable
    execute_pageable(message, options);
    // execute pinned
    execute_pinned(message, options);
    // reset
    hipDeviceReset();
    // exit
    exit(EXIT_SUCCESS);
}