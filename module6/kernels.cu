#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// RANDOM_MAX ensures 0 <= result < 4 to meet the requirement
#define RANDOM_MAX 4
#define KERNEL_STRING "    Kernel"
#define TRANSFER_STRING "    Transfer"
/**
 * Kernel to initial hiprand states
 * Derived from https://docs.nvidia.com/cuda/hiprand
 */
__global__
void initRandoms(unsigned int seed, hiprandState_t* states)
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    hiprand_init(seed, thread_idx, 0, &states[thread_idx]);
}
/**
 * Kernel to initialize randoms using hiprand states 
 * Derived from https://docs.nvidia.com/cuda/hiprand
 */
__global__
void randoms(hiprandState_t* states, unsigned int* result)
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int random = hiprand(&states[thread_idx]) % RANDOM_MAX; 
    result[thread_idx] = random;
}
/**
 * Kernel to generate a sequence of numbers
 */
__global__
void sequence(unsigned int* result)
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    result[thread_idx] = thread_idx;
}
/**
 * Kernel to add two arrays and put the sums in a third array
 */
__global__
void add(unsigned int* result, unsigned int* sequence, unsigned int* random)
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int sum = sequence[thread_idx] + random[thread_idx];
    result[thread_idx] = sum;
}
/**
 * Kernel to subtract two arrays and put the differences in a third array
 */
__global__
void subtract(unsigned int * result, unsigned int* sequence, unsigned int* random)
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int difference = sequence[thread_idx] - random[thread_idx];
    result[thread_idx] = difference;
}
/**
 * Kernel to multiply two arrays and put the product in a third array
 */
__global__
void mult(unsigned int * result, unsigned int* sequence, unsigned int* random)
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int product = sequence[thread_idx] * random[thread_idx];
    result[thread_idx] = product;
}
/**
 * Kernel to modulus two arrays and put the remainder in a third array
 */
__global__
void mod(unsigned int * result, unsigned int* sequence, unsigned int* random)
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int modulus = sequence[thread_idx] % random[thread_idx]; 
    result[thread_idx] = modulus;
}
/**
 * Setup Timer
 *
 * start - start marker
 * stop - stop marker
 */
void setupTimer(hipEvent_t* start, hipEvent_t* stop){
    hipEventCreate(start);
    hipEventCreate(stop);
}
/**
 * Log the computed time
 * 
 * start - start marker
 * stop - stop marker
 * message - log message
 */
void logTime(hipEvent_t start, hipEvent_t stop, const char* message){
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("  %8s: %f\n", message, elapsed);
}
/**
 *  Clean up memory for timers
 *
 * start - start marker
 * stop - stop marker
 */
void cleanTimer(hipEvent_t start, hipEvent_t stop){
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
/**
 * Generate Sequence Array
 *
 * results - results structure
 * device - device structure
 * options - cli structure
 * message - log message
 */
void generateSequence(Results* results, 
                      Device device,
                      Options options, 
                      const char* message) 
{
    // markers
    hipEvent_t start, stop;
    printf("%sSequence:\n", message);
    // setup timer
    setupTimer(&start, &stop);
    // mart start
    hipEventRecord(start, 0);
    // execute the sequence kernel
    sequence<<<options.totalBlocks, options.blockSize>>>(device.block);
    // log time
    logTime(start, stop, KERNEL_STRING);
    // sync
    hipDeviceSynchronize();
    // check error
    hipGetLastError();
    hipEventRecord(start, 0);
    // store generated sequences
    hipMemcpy(results->sequence,
               device.block,
               options.arraySize,
               hipMemcpyDeviceToHost);
    hipMemcpy(device.sequence,
               device.block,
               options.arraySize,
               hipMemcpyDeviceToHost);
    //log time
    logTime(start, stop, TRANSFER_STRING);
    //cleanup timer
    cleanTimer(start, stop);
}
/**
 * Generate Random Array
 *
 * results - results structure
 * device - device structure
 * options - cli structure
 * message - log message
 */
void generateRandom(Results* results, 
                    Device device, 
                    Options options, 
                    const char* message) 
{
    // markers
    hipEvent_t start, stop;
    printf("%sRandom:\n", message);
    // setup timer
    setupTimer(&start, &stop);
    hipMalloc((void**)&device.states,
                options.totalThreads * sizeof(hiprandState_t));
    // mark time
    hipEventRecord(start, 0);
    // execute the sequence kernel
    initRandoms<<<options.totalBlocks, options.blockSize>>>(time(0), device.states);
    randoms<<<options.totalBlocks, options.blockSize>>>(device.states, device.block);
    // log time
    logTime(start, stop, KERNEL_STRING);
    // sync
    hipDeviceSynchronize();
    // check error
    hipGetLastError();
    hipEventRecord(start, 0);
    // store generated sequences
    hipMemcpy(results->random,
               device.block,
               options.arraySize,
               hipMemcpyDeviceToHost);
    hipMemcpy(device.random,
               device.block,
               options.arraySize,
               hipMemcpyDeviceToHost);
    // log time
    logTime(start, stop, TRANSFER_STRING);
    // clean timer
    cleanTimer(start, stop);
}
/**
 * Perform addition
 *
 * results - results structure
 * device - device structure
 * options - cli structure
 */
void performAdd(Results* results, Device device, Options options)
{
    // markers
    hipEvent_t start, stop;
    printf("  Adding\n");
    // set up timer
    setupTimer(&start, &stop);
    // mark time
    hipEventRecord(start, 0);
    // execute the sequence kernel
    add<<<options.totalBlocks, options.blockSize>>>(device.block,
                                                    device.sequence,
                                                    device.random);
    // log time
    logTime(start, stop, KERNEL_STRING);
    // sync
    hipDeviceSynchronize();
    // check error
    hipGetLastError();
    hipEventRecord(start, 0);
    // store generated sequences
    hipMemcpy(results->sum, 
               device.block, 
               options.arraySize,
               hipMemcpyDeviceToHost);
    // log time
    logTime(start, stop, TRANSFER_STRING);
    // clean up
    cleanTimer(start, stop);
}
/**
 * Perform subtraction
 *
 * results - results structure
 * device - device structure
 * options - cli structure
 */
void performSubtract(Results* results, Device device, Options options)
{
    // markers
    hipEvent_t start, stop;
    printf("  Subtracting\n");
    setupTimer(&start, &stop);
    // mark time
    hipEventRecord(start, 0);
    // execute the sequence kernel
    subtract<<<options.totalBlocks, options.blockSize>>>(device.block,
                                                    device.sequence,
                                                    device.random);
    // log time
    logTime(start, stop, KERNEL_STRING);
    // sync
    hipDeviceSynchronize();
    // check error
    hipGetLastError();
    hipEventRecord(start, 0);
    // store generated sequences
    hipMemcpy(results->difference, 
               device.block, 
               options.arraySize,
               hipMemcpyDeviceToHost);
    // log time
    logTime(start, stop, TRANSFER_STRING);
    // clean up
    cleanTimer(start, stop);
}
/**
 * Perform modulus
 *
 * results - results structure
 * device - device structure
 * options - cli structure
 */
void performModulus(Results* results, Device device, Options options)
{
    // marker
    hipEvent_t start, stop;
    printf("  Modulus\n");
    setupTimer(&start, &stop);
    // mark time
    hipEventRecord(start, 0);
    // execute the sequence kernel
    mod<<<options.totalBlocks, options.blockSize>>>(device.block,
                                                    device.sequence,
                                                    device.random);
    // log time
    logTime(start, stop, KERNEL_STRING);
    // sync
    hipDeviceSynchronize();
    // check error
    hipGetLastError();
    hipEventRecord(start, 0);
    // store generated sequences
    hipMemcpy(results->modulus, 
               device.block, 
               options.arraySize,
               hipMemcpyDeviceToHost);
    // log time
    logTime(start, stop, TRANSFER_STRING);
    // clean up
    cleanTimer(start, stop);
}
/**
 * Perform multiple
 *
 * results - results structure
 * device - device structure
 * options - cli structure
 */
void performMult(Results* results, Device device, Options options)
{
    // array sizes
    hipEvent_t start, stop;
    printf("  Multiplying\n");
    setupTimer(&start, &stop);
    // mark time
    hipEventRecord(start, 0);
    // execute the sequence kernel
    mult<<<options.totalBlocks, options.blockSize>>>(device.block,
                                                    device.sequence,
                                                    device.random);
    // log time
    logTime(start, stop, KERNEL_STRING);
    // sync
    hipDeviceSynchronize();
    // check error
    hipGetLastError();
    hipEventRecord(start, 0);
    // store generated sequences
    hipMemcpy(results->product, 
               device.block, 
               options.arraySize,
               hipMemcpyDeviceToHost);
    // log time
    logTime(start, stop, TRANSFER_STRING);
    // clean up timers
    cleanTimer(start, stop);
}
/**
 * Perform addition
 *
 * results - results structure
 * device - device structure
 * options - cli structure
 * message - log message
 */
void perform(Results results, 
             Device device, 
             Options options, 
             const char * message)
{
    printf("%s\n  Generate:\n", message);
    // generate
    generateSequence(&results, device, options, "    ");
    generateRandom(&results, device, options, "    ");
    // perform
    performAdd(&results, device, options);
    performSubtract(&results, device, options);
    performMult(&results, device, options);
    performModulus(&results, device, options);
}
