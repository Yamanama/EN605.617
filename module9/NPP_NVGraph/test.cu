#include <iostream>
#include <hip/hip_runtime.h>
#include <npp.h>
#include <opencv2/opencv.hpp>

using namespace std;

int main()
{    
    const int width = 640, height = 480;

    //Create an 8 bit single channel image
    IplImage* img = cvCreateImage(cvSize(width,height),IPL_DEPTH_8U,1);
    //Set All Image Pixels To 0
    cvZero(img);

    cvShowImage("Input",img);
    cvWaitKey();


    const int step = img->widthStep;
    const int bytes = img->widthStep * img->height;

    unsigned char *dSrc, *dDst;
    hipMalloc<unsigned char>(&dSrc,bytes);
    hipMalloc<unsigned char>(&dDst,bytes);

    //Copy Data From IplImage to Device Pointer
    hipMemcpy(dSrc,img->imageData,bytes,hipMemcpyHostToDevice);

    NppiSize size;
    size.width = width;
    size.height = height;

    const Npp8u value = 150;

    //Call NPP function to add a constant value to each pixel of the image
    nppiAddC_8u_C1RSfs(dSrc,step,value,dDst,step,size,1);

    //Copy back the result from device to IplImage
    hipMemcpy(img->imageData,dDst,bytes,hipMemcpyDeviceToHost);

    hipFree(dSrc);
    hipFree(dDst);

    cvShowImage("Output",img);
    cvWaitKey();

    cvReleaseImage(&img);

    return 0;
}
